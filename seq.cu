#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

void task1(void)
{
    const int N = 50000;
    int sum = 0, sumA = 0, i = 0;

    thrust::device_vector<int>a(N);
    thrust::sequence(a.begin(), a.end(), 0);
    sumA = thrust::reduce(a.begin(), a.end(), 0);

    for (; i < N; i ++) {
        sum += i;
    }
    std::cout << sum << std::endl;
    std::cout << sumA << std::endl;
}

__global__ void fillKernel(int *a, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        a[tid] = tid;
    }
}

void fill(int* d_a, int n)
{
    int block = 512;
    int nblock = n / block + (( n % block) ? 1 : 0);

    fillKernel <<< nblock, block >>> (d_a, n);
}

void task2(void)
{
    const int N = 50000;
    int sum = 0, sumA = 0, i = 0;

    thrust::device_vector<int>a(N);
    fill(thrust::raw_pointer_cast(&a[0]), N);
    sumA = thrust::reduce(a.begin(), a.end(), 0);

    for (; i < N; i ++) {
        sum += i;
    }
    std::cout << sum << std::endl;
    std::cout << sumA << std::endl;
}

void task3(void)
{
    const int N = 50000;
    int sum = 0, sumA = 0, i = 0;

    thrust::device_vector<int>a(N);
    thrust::sequence(a.begin(), a.end(), 0);

    #pragma omp parallel for reduction(+ : sum)
    for (i = 0; i < N; i ++) sum += i;

    sumA = thrust::reduce(a.begin(), a.end(), 0);

    std::cout << sum << std::endl;
    std::cout << sumA << std::endl;
}

int main(void)
{
    task1();
    task2();
    task3();
}